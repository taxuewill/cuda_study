#include "hip/hip_runtime.h"
#include "common.h"

#include <stdio.h>


__global__ void
cudaVectorAdd(const int *A, const int *B, int *C, int numElements)
{
    // int i = threadIdx.x;
    // printf("add %d",blockDim.x);
    // if (i < numElements)
    // {
    //     C[i] = A[i] + B[i];
    // }
    printf("CUDA!\n");
}


void vectorAdd(const int *A,const int *B,int * C,int numElements){
    printf("vectorAdd\n");
    
    cudaVectorAdd<<<1,30>>>(A,B,C,numElements);
}